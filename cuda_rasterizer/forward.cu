#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "forward.h"
#include "auxiliary.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

// Forward method for converting the input spherical harmonics
// coefficients of each Gaussian to a simple RGB color.
__device__ glm::vec3 computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs, bool* clamped)
{
	// The implementation is loosely based on code for 
	// "Differentiable Point-Based Radiance Fields for 
	// Efficient View Synthesis" by Zhang et al. (2022)
	glm::vec3 pos = means[idx];
	glm::vec3 dir = pos - campos;
	dir = dir / glm::length(dir);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;
	glm::vec3 result = SH_C0 * sh[0];

	if (deg > 0)
	{
		float x = dir.x;
		float y = dir.y;
		float z = dir.z;
		result = result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;
			result = result +
				SH_C2[0] * xy * sh[4] +
				SH_C2[1] * yz * sh[5] +
				SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
				SH_C2[3] * xz * sh[7] +
				SH_C2[4] * (xx - yy) * sh[8];

			if (deg > 2)
			{
				result = result +
					SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
					SH_C3[1] * xy * z * sh[10] +
					SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
					SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[12] +
					SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
					SH_C3[5] * z * (xx - yy) * sh[14] +
					SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
			}
		}
	}
	result += 0.5f;

	// RGB colors are clamped to positive values. If values are
	// clamped, we need to keep track of this for the backward pass.
	clamped[3 * idx + 0] = (result.x < 0);
	clamped[3 * idx + 1] = (result.y < 0);
	clamped[3 * idx + 2] = (result.z < 0);
	return glm::max(result, 0.0f);
}

// Forward version of 2D covariance matrix computation
__device__ float3 computeCov2D(const float3& mean, float focal_x, float focal_y, float tan_fovx, float tan_fovy, const float* cov3D, const float* viewmatrix)
{
	// The following models the steps outlined by equations 29
	// and 31 in "EWA Splatting" (Zwicker et al., 2002). 
	// Additionally considers aspect / scaling of viewport.
	// Transposes used to account for row-/column-major conventions.
	float3 t = transformPoint4x3(mean, viewmatrix);

	const float limx = 1.3f * tan_fovx;
	const float limy = 1.3f * tan_fovy;
	const float txtz = t.x / t.z;
	const float tytz = t.y / t.z;
	t.x = min(limx, max(-limx, txtz)) * t.z;
	t.y = min(limy, max(-limy, tytz)) * t.z;

	glm::mat3 J = glm::mat3(
		focal_x / t.z, 0.0f, -(focal_x * t.x) / (t.z * t.z),
		0.0f, focal_y / t.z, -(focal_y * t.y) / (t.z * t.z),
		0, 0, 0);

	glm::mat3 W = glm::mat3(
		viewmatrix[0], viewmatrix[4], viewmatrix[8],
		viewmatrix[1], viewmatrix[5], viewmatrix[9],
		viewmatrix[2], viewmatrix[6], viewmatrix[10]);

	glm::mat3 T = W * J;

	glm::mat3 Vrk = glm::mat3(
		cov3D[0], cov3D[1], cov3D[2],
		cov3D[1], cov3D[3], cov3D[4],
		cov3D[2], cov3D[4], cov3D[5]);

	glm::mat3 cov = glm::transpose(T) * glm::transpose(Vrk) * T;

	// Apply low-pass filter: every Gaussian should be at least
	// one pixel wide/high. Discard 3rd row and column.
	cov[0][0] += 0.3f;
	cov[1][1] += 0.3f;
	return { float(cov[0][0]), float(cov[0][1]), float(cov[1][1]) };
}

// Forward method for converting scale and rotation properties of each
// Gaussian to a 3D covariance matrix in world space. Also takes care
// of quaternion normalization.
__device__ void computeCov3D(const glm::vec3 scale, float mod, const glm::vec4 rot, float* cov3D)
{
	// Create scaling matrix
	glm::mat3 S = glm::mat3(1.0f);
	S[0][0] = mod * scale.x;
	S[1][1] = mod * scale.y;
	S[2][2] = mod * scale.z;

	// Normalize quaternion to get valid rotation
	glm::vec4 q = rot;// / glm::length(rot);
	float r = q.x;
	float x = q.y;
	float y = q.z;
	float z = q.w;

	// Compute rotation matrix from quaternion
	glm::mat3 R = glm::mat3(
		1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
		2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
		2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
	);

	glm::mat3 M = S * R;

	// Compute 3D world covariance matrix Sigma
	glm::mat3 Sigma = glm::transpose(M) * M;

	// Covariance is symmetric, only store upper right
	cov3D[0] = Sigma[0][0];
	cov3D[1] = Sigma[0][1];
	cov3D[2] = Sigma[0][2];
	cov3D[3] = Sigma[1][1];
	cov3D[4] = Sigma[1][2];
	cov3D[5] = Sigma[2][2];
}

// Perform initial steps for each Gaussian prior to rasterization.
template<int C>
__global__ void preprocessCUDA(int P, int D, int M,
	const float* orig_points,
	const glm::vec3* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* shs,
	bool* clamped,
	const float* cov3D_precomp,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, int H,
	const float tan_fovx, float tan_fovy,
	const float focal_x, float focal_y,
	int* radii,
	float2* points_xy_image,
	float* depths,
	float* cov3Ds,
	float* rgb,
	float4* conic_opacity,
	const dim3 grid,
	uint32_t* tiles_touched,
	bool prefiltered,
	const int tile_size)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Initialize radius and touched tiles to 0. If this isn't changed,
	// this Gaussian will not be processed further.
	radii[idx] = 0;
	tiles_touched[idx] = 0;

	// Perform near culling, quit if outside.
	float3 p_view;
	if (!in_frustum(idx, orig_points, viewmatrix, projmatrix, prefiltered, p_view))
		return;

	// Transform point by projecting
	float3 p_orig = { orig_points[3 * idx], orig_points[3 * idx + 1], orig_points[3 * idx + 2] };
	float4 p_hom = transformPoint4x4(p_orig, projmatrix);
	float p_w = 1.0f / (p_hom.w + 0.0000001f);
	float3 p_proj = { p_hom.x * p_w, p_hom.y * p_w, p_hom.z * p_w };

	// If 3D covariance matrix is precomputed, use it, otherwise compute
	// from scaling and rotation parameters. 
	const float* cov3D;
	if (cov3D_precomp != nullptr)
	{
		cov3D = cov3D_precomp + idx * 6;
	}
	else
	{
		computeCov3D(scales[idx], scale_modifier, rotations[idx], cov3Ds + idx * 6);
		cov3D = cov3Ds + idx * 6;
	}

	// Compute 2D screen-space covariance matrix
	float3 cov = computeCov2D(p_orig, focal_x, focal_y, tan_fovx, tan_fovy, cov3D, viewmatrix);

	// Invert covariance (EWA algorithm)
	float det = (cov.x * cov.z - cov.y * cov.y);
	if (det == 0.0f)
		return;
	float det_inv = 1.f / det;
	float3 conic = { cov.z * det_inv, -cov.y * det_inv, cov.x * det_inv };

	// ---- OctagonSplat: Start of new culling logic ----

	// 1. Calculate t and check for visibility
	float t = 2.f * log(255.f * opacities[idx]);
	if (t < 0.0f) {
		// Gaussian is too transparent to be visible, cull it.
		return;
	}

	float a = conic.x;
	float b = conic.y;
	float c = conic.z;

	// 2. Calculate the 8 vertices of the octagon
	// V0, V1 are for slope m = -1
	// V2, V3 are for slope inf
	// V4, V5 are for slope 1
	// V6, V7 are for slope 0
	float2 V[8];
	
	// Pre-calculate reused terms
	float b2 = b * b;
	
	// m = 1 & m = -1 (diagonal vertices)
	float d1 = a + 2*b + c;
	if (d1 <= 0.f) return;
	float x1 = sqrtf(t * (b+c)*(b+c) / (d1 * (a*c - b2)));
    V[4].x = x1; V[4].y = -(a+b)/(b+c) * x1;
    V[5].x = -x1; V[5].y = -V[4].y;

	float d2 = a - 2*b + c;
    if (d2 <= 0.f) return;
    float x2 = sqrtf(t * (c-b)*(c-b) / (d2 * (a*c - b2)));
    V[0].x = x2; V[0].y = -(a-b)/(c-b) * x2;
    V[1].x = -x2; V[1].y = -V[0].y;

	// m = 0 & m = inf (axis-aligned vertices)
    // Handle potential division by zero
    if (c == 0.f) return;
    float x_m_inf = sqrtf(t * c / (a*c - b2));
    V[2].x = x_m_inf; V[2].y = -b/c * x_m_inf;
    V[3].x = -x_m_inf; V[3].y = -V[2].y;
    
    if (a == 0.f) return;
    float y_m_0 = sqrtf(t * a / (a*c - b2));
    V[6].y = y_m_0; V[6].x = -b/a * y_m_0;
    V[7].y = -y_m_0; V[7].x = -V[6].x;


	float2 point_image = { ndc2Pix(p_proj.x, W), ndc2Pix(p_proj.y, H) };

	// Add center offset to all vertices
	#pragma unroll
	for(int i = 0; i < 8; ++i) {
		V[i].x += point_image.x;
		V[i].y += point_image.y;
	}

	// 3. Coarse Culling: Get AABB of the octagon
	float2 min_coord = V[0], max_coord = V[0];
	#pragma unroll
	for(int i = 1; i < 8; ++i) {
		min_coord.x = min(min_coord.x, V[i].x);
		min_coord.y = min(min_coord.y, V[i].y);
		max_coord.x = max(max_coord.x, V[i].x);
		max_coord.y = max(max_coord.y, V[i].y);
	}

	// Convert AABB to tile coordinates
	int min_tile_x = max(0, (int)floor(min_coord.x / tile_size));
	int max_tile_x = min((int)grid.x - 1, (int)floor(max_coord.x / tile_size));
	int min_tile_y = max(0, (int)floor(min_coord.y / tile_size));
	int max_tile_y = min((int)grid.y - 1, (int)floor(max_coord.y / tile_size));

	if (max_tile_x < min_tile_x || max_tile_y < min_tile_y) {
        return;
    }
	
	// 4. Fine Culling: Edge function tests for tiles in the AABB
	uint32_t tiles_count = 0;
	for (int ty = min_tile_y; ty <= max_tile_y; ++ty) {
		for (int tx = min_tile_x; tx <= max_tile_x; ++tx) {
			float2 tile_corners[4] = {
				{(float)tx * tile_size, (float)ty * tile_size},
				{(float)(tx+1) * tile_size, (float)ty * tile_size},
				{(float)tx * tile_size, (float)(ty+1) * tile_size},
				{(float)(tx+1) * tile_size, (float)(ty+1) * tile_size}
			};

			bool tile_is_outside = false;
			// Check against each of the 8 edges of the octagon
			#pragma unroll
			for (int i = 0; i < 8; ++i) {
				float2 v1 = V[i];
				float2 v2 = V[(i + 1) % 8];
				
				// Check if all 4 corners of the tile are "outside" this edge
				int out_count = 0;
				#pragma unroll
				for (int j = 0; j < 4; ++j) {
					// Edge function
					float edge_val = (tile_corners[j].x - v1.x) * (v2.y - v1.y) - (tile_corners[j].y - v1.y) * (v2.x - v1.x);
					if (edge_val > 0) {
						out_count++;
					}
				}

				if (out_count == 4) {
					// All corners are outside this edge, so the tile does not intersect
					tile_is_outside = true;
					break; // No need to check other edges
				}
			}

			if (!tile_is_outside) {
				// This tile intersects the octagon
				tiles_count++;
			}
		}
	}

	if (tiles_count == 0) {
		return;
	}

	// ---- OctagonSplat: End of new culling logic ----

	// If colors have been precomputed, use them, otherwise convert
	// spherical harmonics coefficients to RGB color.
	if (colors_precomp == nullptr)
	{
		glm::vec3 result = computeColorFromSH(idx, D, M, (glm::vec3*)orig_points, *cam_pos, shs, clamped);
		rgb[idx * C + 0] = result.x;
		rgb[idx * C + 1] = result.y;
		rgb[idx * C + 2] = result.z;
	}

	// Store some useful helper data for the next steps.
	depths[idx] = p_view.z;
    radii[idx] = 0; // Radius is no longer used for culling
	points_xy_image[idx] = point_image;
	// Inverse 2D covariance and opacity neatly pack into one float4
	conic_opacity[idx] = {conic.x, conic.y, conic.z, opacities[idx]};
    tiles_touched[idx] = tiles_count;
}

// Main rasterization method. Collaboratively works on one tile per
// block, each thread treats one pixel. Alternates between fetching 
// and rasterizing data.
template <uint32_t CHANNELS>
__global__ void renderCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	const float2* __restrict__ points_xy_image,
	const float* __restrict__ features,
	const float4* __restrict__ conic_opacity,
	float* __restrict__ final_T,
	uint32_t* __restrict__ n_contrib,
	const float* __restrict__ bg_color,
	float* __restrict__ out_color,
	const int tile_size)
{
	// Identify current tile and associated min/max pixel range.
	auto block = cg::this_thread_block();
	uint32_t horizontal_blocks = (W + tile_size - 1) / tile_size;
	uint2 pix_min = { block.group_index().x * tile_size, block.group_index().y * tile_size };
	uint2 pix_max = { min(pix_min.x + tile_size, W), min(pix_min.y + tile_size , H) };
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	uint32_t pix_id = W * pix.y + pix.x;
	float2 pixf = { (float)pix.x, (float)pix.y };

	// Check if this thread is associated with a valid pixel or outside.
	bool inside = pix.x < W&& pix.y < H;
	// Done threads can help with fetching, but don't rasterize
	bool done = !inside;

	// Load start/end range of IDs to process in bit sorted list.
	uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
	const int block_size = tile_size * tile_size;  // 动态计算block大小
	const int rounds = ((range.y - range.x + block_size - 1) / block_size);
	int toDo = range.y - range.x;

	// Allocate storage for batches of collectively fetched data.
	// 注意：这里需要使用动态共享内存或者预分配最大的可能尺寸
	extern __shared__ char shared_mem[];
	int* collected_id = (int*)shared_mem;
	float2* collected_xy = (float2*)(collected_id + block_size);
	float4* collected_conic_opacity = (float4*)(collected_xy + block_size);

	// Initialize helper variables
	float T = 1.0f;
	uint32_t contributor = 0;
	uint32_t last_contributor = 0;
	float C[CHANNELS] = { 0 };

	// Iterate over batches until all done or range is complete
	for (int i = 0; i < rounds; i++, toDo -= block_size)
	{
		// End if entire block votes that it is done rasterizing
		int num_done = __syncthreads_count(done);
		if (num_done == block_size)
			break;

		// Collectively fetch per-Gaussian data from global to shared
		int progress = i * block_size + block.thread_rank();
		if (range.x + progress < range.y)
		{
			int coll_id = point_list[range.x + progress];
			collected_id[block.thread_rank()] = coll_id;
			collected_xy[block.thread_rank()] = points_xy_image[coll_id];
			collected_conic_opacity[block.thread_rank()] = conic_opacity[coll_id];
		}
		block.sync();

		// Iterate over current batch
		for (int j = 0; !done && j < min(block_size, toDo); j++)
		{
			// Keep track of current position in range
			contributor++;

			// Resample using conic matrix (cf. "Surface 
			// Splatting" by Zwicker et al., 2001)
			float2 xy = collected_xy[j];
			float2 d = { xy.x - pixf.x, xy.y - pixf.y };
			float4 con_o = collected_conic_opacity[j];
			float power = -0.5f * (con_o.x * d.x * d.x + con_o.z * d.y * d.y) - con_o.y * d.x * d.y;
			if (power > 0.0f)
				continue;

			// Eq. (2) from 3D Gaussian splatting paper.
			// Obtain alpha by multiplying with Gaussian opacity
			// and its exponential falloff from mean.
			// Avoid numerical instabilities (see paper appendix). 
			float alpha = min(0.99f, con_o.w * exp(power));
			if (alpha < 1.0f / 255.0f)
				continue;
			float test_T = T * (1 - alpha);
			if (test_T < 0.0001f)
			{
				done = true;
				continue;
			}

			// Eq. (3) from 3D Gaussian splatting paper.
			for (int ch = 0; ch < CHANNELS; ch++)
				C[ch] += features[collected_id[j] * CHANNELS + ch] * alpha * T;

			T = test_T;

			// Keep track of last range entry to update this
			// pixel.
			last_contributor = contributor;
		}
	}

	// All threads that treat valid pixel write out their final
	// rendering data to the frame and auxiliary buffers.
	if (inside)
	{
		final_T[pix_id] = T;
		n_contrib[pix_id] = last_contributor;
		for (int ch = 0; ch < CHANNELS; ch++)
			out_color[ch * H * W + pix_id] = C[ch] + T * bg_color[ch];
	}
}

void FORWARD::render(
	const dim3 grid, dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	int W, int H,
	const float2* means2D,
	const float* colors,
	const float4* conic_opacity,
	float* final_T,
	uint32_t* n_contrib,
	const float* bg_color,
	float* out_color,
	const int tile_size)
{
	// 计算动态共享内存大小
	const int block_size = tile_size * tile_size;
	const size_t shared_mem_size = block_size * (sizeof(int) + sizeof(float2) + sizeof(float4));
	
	renderCUDA<NUM_CHANNELS> << <grid, block, shared_mem_size >> > (
		ranges,
		point_list,
		W, H,
		means2D,
		colors,
		conic_opacity,
		final_T,
		n_contrib,
		bg_color,
		out_color,
		tile_size);
}

void FORWARD::preprocess(int P, int D, int M,
	const float* means3D,
	const glm::vec3* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* shs,
	bool* clamped,
	const float* cov3D_precomp,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, int H,
	const float focal_x, float focal_y,
	const float tan_fovx, float tan_fovy,
	int* radii,
	float2* means2D,
	float* depths,
	float* cov3Ds,
	float* rgb,
	float4* conic_opacity,
	const dim3 grid,
	uint32_t* tiles_touched,
	bool prefiltered,
	const int tile_size)
{
	preprocessCUDA<NUM_CHANNELS> << <(P + 255) / 256, 256 >> > (
		P, D, M,
		means3D,
		scales,
		scale_modifier,
		rotations,
		opacities,
		shs,
		clamped,
		cov3D_precomp,
		colors_precomp,
		viewmatrix, 
		projmatrix,
		cam_pos,
		W, H,
		tan_fovx, tan_fovy,
		focal_x, focal_y,
		radii,
		means2D,
		depths,
		cov3Ds,
		rgb,
		conic_opacity,
		grid,
		tiles_touched,
		prefiltered,
		tile_size
		);
}
